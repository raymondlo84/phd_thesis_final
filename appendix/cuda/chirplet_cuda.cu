#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <hipblas.h>
//#include <cutil.h>
//#include <cutil_inline.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include "chirplet_cuda.h"
#define cutilSafeCall(x) checkCudaErrors(x)
#define cutilCheckMsg(x) getLastCudaError(x)
#define cutStartTimer(x) sdkStartTimer(x)
#define cutCreateTimer(x) sdkCreateTimer(x)
#define cutStopTimer(x) sdkStopTimer(x)
#define cutGetTimerValue(x) sdkGetTimerValue(x)
#define cutResetTimer(x) sdkResetTimer(x)
#define cutDeleteTimer(x) sdkDeleteTimer(x)
__constant__ float sqrtOfPI = 1.77245385;

//**************************************************************************
__constant__ float muB = 11.823222f;
__constant__ float muG = 16.465048f;
__constant__ float muR = 18.527411f;

__constant__ float histRes = (float)histogramSize;
__constant__ int cdf_samples_per_thread = CDFSAMPLES;
__constant__ int cdf_threads = CDFSMEM;
//**************************************************************************

//unsigned int htimer;
StopWatchInterface *htimer = NULL;
long imgsize;

unsigned char* d_in;
unsigned char* d_output;


float* d_res;
float* d_weight;
float* d_light;
float* d_log_light;
float* d_log_light_lowpassed;
float* d_log_normalized;
float* d_log_normalized_transposed;
float* d_H;
float* d_V;
float* d_blur;
float* d_blur_transposed;
float* d_detail;
float* d_detail_domains;

float* h_log_light;

hipError_t cudaStat ;
hipblasStatus_t stat ;
hipblasHandle_t handle;

int _tpb;
int _tpbh;
int _bpg;
int _bpgc;
int _bpgh;

int _wid;
int _hi;

float _q_scalar;
int _num_img;

void hdrInit(int wid, int hi, int colrDep, unsigned int num_img){
	hipDeviceReset();
	imgsize = wid*hi*colrDep;
	_tpb = TPB;
	_tpbh = TPB_HIST;//TPB;//histogramSize;
	_bpg = (wid*hi + _tpb - 1)/_tpb;
	_bpgc = (imgsize + _tpb - 1)/_tpb;
	_bpgh = (imgsize + _tpbh*PARTIAL_HIST - 1)/(_tpbh*PARTIAL_HIST);
	_wid = wid;
	_hi = hi;
	_num_img = num_img;
	cutCreateTimer (&htimer);
	int i;
	
	cutilSafeCall(hipMalloc((void**)&d_blur, _wid*_hi*sizeof(float)));
	cutilSafeCall(hipMemset((void*)d_blur,0,_wid*_hi*sizeof(float)));

	cutilSafeCall(hipMalloc((void**)&d_detail, _wid*_hi*NUM_DOMAINS*sizeof(float)));
	cutilSafeCall(hipMemset((void*)d_detail,0,_wid*_hi*NUM_DOMAINS*sizeof(float)));

	cutilSafeCall(hipMalloc((void**)&d_detail_domains, _wid*_hi*NUM_DOMAINS*sizeof(float)));
	cutilSafeCall(hipMemset((void*)d_detail_domains,0,_wid*_hi*NUM_DOMAINS*sizeof(float)));

	cutilSafeCall(hipMalloc((void**)&d_blur_transposed, _wid*_hi*sizeof(float)));
	cutilSafeCall(hipMemset((void*)d_blur_transposed,0,_wid*_hi*sizeof(float)));
	
	cutilSafeCall(hipMalloc((void**)&d_in, imgsize*_num_img*sizeof(unsigned char)));
	cutilSafeCall(hipMemset((void*)d_in,0,imgsize*_num_img*sizeof(unsigned char)));
	
	cutilSafeCall(hipMalloc((void**)&d_light, imgsize*sizeof(float)));
	cutilSafeCall(hipMemset((void*)d_light,0,imgsize*sizeof(float)));

	cutilSafeCall(hipMalloc((void**)&d_log_light, _wid*_hi*sizeof(float)));
	cutilSafeCall(hipMemset((void*)d_log_light,0,_wid*_hi*sizeof(float)));

	cutilSafeCall(hipMalloc((void**)&d_log_light_lowpassed, _wid*_hi*sizeof(float)));
	cutilSafeCall(hipMemset((void*)d_log_light_lowpassed,0,_wid*_hi*sizeof(float)));

	cutilSafeCall(hipMalloc((void**)&d_log_normalized, _wid*_hi*sizeof(float)));
	cutilSafeCall(hipMemset((void*)d_log_normalized,0,_wid*_hi*sizeof(float)));

	cutilSafeCall(hipMalloc((void**)&d_log_normalized_transposed, _wid*_hi*sizeof(float)));
	cutilSafeCall(hipMemset((void*)d_log_normalized_transposed,0,_wid*_hi*sizeof(float)));
	
	cutilSafeCall(hipMalloc((void**)&d_H, _wid*_hi*sizeof(float)));
	cutilSafeCall(hipMemset((void*)d_H,0,_wid*_hi*sizeof(float)));
	
	cutilSafeCall(hipMalloc((void**)&d_V, _wid*_hi*sizeof(float)));
	cutilSafeCall(hipMemset((void*)d_V,0,_wid*_hi*sizeof(float)));
	
	cutilSafeCall(hipMalloc((void**)&d_output, imgsize*sizeof(unsigned char)));
	cutilSafeCall(hipMemset((void*)d_output,0,imgsize*sizeof(unsigned char)));
	
	cutilSafeCall(hipMalloc((void**)&d_res, PIX_DEP*colrDep*sizeof(float)));
	cutilSafeCall(hipMemset((void*)d_res,0,PIX_DEP*colrDep*sizeof(float)));
	
	cutilSafeCall(hipMalloc((void**)&d_weight, PIX_DEP*num_img*sizeof(float)));
	cutilSafeCall(hipMemset((void*)d_weight,0,PIX_DEP*num_img*sizeof(float)));
	
	h_log_light = (float*)malloc(_wid*_hi*sizeof(float));

	stat = hipblasCreate(&handle);
}

void RIW_CopyToDevice(float* input_res, float* input_weight, float q_scalar){
	int tmpsize = PIX_DEP;
	//printf("copying %d\n",tmpsize);
	cutStartTimer(&htimer);
	_q_scalar = q_scalar;
	cutilSafeCall(hipMemcpy(d_res, input_res, PIX_DEP*3*sizeof(float), hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(d_weight, input_weight, PIX_DEP*_num_img*sizeof(float), hipMemcpyHostToDevice));
	cutStopTimer(&htimer);
	//printf ("transfer to device ms time: %f ms\n", cutGetTimerValue(htimer));
	cutResetTimer(&htimer);
}

void frameCopyToDevice(unsigned char* input, int index){
	cutStartTimer(&htimer);
	cutilSafeCall(hipMemcpy(d_in+index*imgsize, input, imgsize*sizeof(unsigned char), hipMemcpyHostToDevice));
	cutStopTimer(&htimer);
	//printf ("transfer to device ms time: %f ms\n", cutGetTimerValue(htimer));
	cutResetTimer(&htimer);
}

void frameCopyToHost(unsigned char* output){
	cutStartTimer(&htimer);
	cutilSafeCall(hipMemcpy(output, d_output, imgsize*sizeof(unsigned char), hipMemcpyDeviceToHost));
	cutStopTimer(&htimer);
	//printf ("transfer to host ms time: %f ms\n", cutGetTimerValue(htimer));
	cutResetTimer(&htimer);
}

__global__ void light_composition(unsigned char* input, float* dRes, float* C, float pref, float q_scalar, unsigned int num_img, float pref_exposure, float* output, unsigned int imgsize){
	unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
	int j, k; //j == rgb, k == bmd
	unsigned char p[NUM_IMG];
	float q[NUM_IMG];
	float c[NUM_IMG];
	float c_hat[NUM_IMG];
	float total;
	float light;
	float hdrPref[NUM_IMG];
	long offset = 0;
	float k_hat;
	if (i < imgsize){
		light = 0.0f;
		total = 0.0f;
		j = i % 3;
		for(k=0;k<num_img;k++){
			p[k] = input[i+offset];
			hdrPref[k] = powf(q_scalar, (float)k-pref_exposure+pref);
			offset += imgsize;
		}
	    
		for(k=0;k<num_img;k++){
    		q[k] = dRes[p[k]*3+j];
    		c[k] = C[p[k]*num_img+k];
		}

    	for(k=0;k<num_img;k++){
		    total += c[k];
    		light += q[k]*c[k]*hdrPref[k];
    	}

		output[i] = light/(total+0.000001f);
	}
}

__global__
void ConvertToLogLuminance_cuda(float* img, float* log_lum, int N){
    int b_i = blockIdx.x * blockDim.x + threadIdx.x;
    int b_ii = b_i * 3;
    int g_i = b_ii + 1;
    int r_i = b_ii + 2;
    float log_brightness=10000; //3 frames

    //float log_brightness=10*10*10*10; 
    if(b_i < N){
        float value = 0.3*img[r_i]+0.6*img[g_i]+0.1*img[b_ii];
        float value_2 = 1.0/value;
        img[b_ii] = 1.0*img[b_ii]*value_2;
        img[g_i] = 1.0*img[g_i]*value_2;
        img[r_i] = 1.0*img[r_i]*value_2;
        //log_lum[b_i] = __powf(value*log_brightness, 1/8.0);
        log_lum[b_i] = __logf(1.0+value*log_brightness)/__logf(2)-1.0;
    }
}

__global__
void NormalizeRange(float* img, float range, float* output, float upper_bound, float lower_bound, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < N){
        output[i] = (img[i] - lower_bound)/(upper_bound - lower_bound)*range;
    }
}

__global__
void NormalizeRange2(float* img, float* img_lowpassed, float range, float* output, unsigned int maxIndex, unsigned int minIndex, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float upper_bound = img_lowpassed[maxIndex];
    float lower_bound = img_lowpassed[minIndex];
    if(upper_bound < 8.5f)
	upper_bound = 8.5f;
    if(i < N){
        output[i] = (img[i] - lower_bound)/(upper_bound - lower_bound)*range;
    }
}

__global__ void transpose(float *odata, float *idata, int width, int height, int dummy)
{
	__shared__ float block[BLOCK_DIM][BLOCK_DIM+1];
	
	// read the matrix tile into shared memory
	unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
	unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;
	if((xIndex < width) && (yIndex < height))
	{
		unsigned int index_in = yIndex * width + xIndex;
		block[threadIdx.y][threadIdx.x] = idata[index_in];
	}

	__syncthreads();

	// write the transposed matrix tile to global memory
	xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
	yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;
	if((xIndex < height) && (yIndex < width))
	{
		unsigned int index_out = yIndex * height + xIndex;
		odata[index_out] = block[threadIdx.x][threadIdx.y];
	}
}

__global__ void gradient_vertical(float* input, float* output, unsigned int width, unsigned int height, unsigned int imgsize){
	unsigned int i = (blockIdx.x*blockDim.x+threadIdx.x);
	unsigned int row = i / width;
	if (i < imgsize){
    		if(row == 0)
        		output[i] = 0.0f;
    		else
        		output[i] = fabs(input[i]+input[i-width]);
	}
}

__global__ void domain_filter_vertical_RGB_noreg(float* img, float* dVdy, float a, float s_div_r, int width, int height, int n){	
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    
	int wid = width;
	int hi = height;
	int rowsize = wid*n;
	int idx_ij = j;

	float tmpH;
	float q_e; 
	float tmpD;
		
	if(j< rowsize){
		for(int i = 0; i < hi; i++){
			tmpD = 1.0f+s_div_r*dVdy[idx_ij];
			if(i>0){
				q_e = img[idx_ij];
				tmpH = q_e + __powf(a, tmpD) * ( tmpH - q_e );
				img[idx_ij] = tmpH;
			}
			else
				tmpH = img[idx_ij];
			idx_ij += rowsize;
		}
		for(int i = hi-1; i >= 0; i--){
			idx_ij -= rowsize;
			if(i<hi-1){
			    q_e = img[idx_ij];
				tmpH = q_e + __powf(a, tmpD) * ( tmpH - q_e );
			    img[idx_ij] = tmpH;
			}
			tmpD = 1.0f+s_div_r*dVdy[idx_ij];
		}

	}
}

__global__
void extract_detail(float* img, float* blur, float* output, unsigned int num_domains, int N){
	unsigned long i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int j;
	if(i < N){
		for(j=0;j<num_domains;j++){
			if(j==0)
				output[i] = img[i]-blur[i];
			else
				output[i] = blur[i-N]-blur[i];
			i += N;
		}
	}
}

__global__
void CompressionAndSaturation(float* img, float* detail, float* J3, float meanB, unsigned char* out_img, int N){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int ii = i*3;
	unsigned int j;
	unsigned long detail_offset = i;
	if(i < N){
		const float brightness = -0.05f;
		float layer[NUM_DOMAINS] = {0.4f,0.3f,0.3f}; //detail _ fine texture
		float sFRGB[3] = {0.55f,0.55f,0.55f};
		const float base_value = 1.0; //mid-tone value (may saturate the high lights)
		float local_min = 0.2f;
		float local_max = 7.0f;
		float range = 1.0f;
		float J3_local = (J3[i]-local_min)/(local_max-local_min)*range;
		float LC = 0.0f;
		float output;
		//saturation factor (0.5 = low) & (1 = high)

		for(j=0;j<NUM_DOMAINS;j++){
			if(J3_local<0.5f)
				layer[j] = J3_local*layer[j]/0.5f;
			if(J3_local>6.0f && J3_local <= 7.0f)
				layer[j] = (7.0f-J3_local)*layer[j]/6.0f;
			if(J3_local>7.0f)
				layer[j] = 0.0f;
			LC += layer[j]*detail[detail_offset];
			detail_offset += N;
		}
		LC += brightness + meanB + base_value * (J3_local - meanB);

		for(j=0;j<3;j++){
			output = __powf(img[ii+j], sFRGB[j]) * LC*255.0f;
			if(output > 255.0f)
				output = 255.0f;
			else if(output < 0.0f)
				output = 0.0f;
			out_img[ii+j] = (unsigned char)output;
		}
	}
}

__global__
void lowpass(float* img, float* img_filter, unsigned int filter, int N){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < N){
		if(filter)
			img_filter[i] = img_filter[i]*0.9f + 0.1f*img[i];
		else
			img_filter[i] = img[i];
	}
}

float avgHDR_HE = 0.0f;
float cnt = 0.0f;
int indexMax;
int indexMin;
float img_mean;

float max_light = 0.0f;
float min_light = 0.0f;


unsigned int run = 1;
unsigned int useLIGHT(float muLight, float exposure_pref){
	cnt += 1.0f;
	
	cutStartTimer(&htimer);
	light_composition <<< _bpgc, _tpb >>> (d_in, d_res, d_weight, muLight, _q_scalar, _num_img, exposure_pref, d_light, imgsize);
	hipDeviceSynchronize();
	
	ConvertToLogLuminance_cuda<<<_bpg, _tpb>>>(d_light, d_log_light, _wid*_hi);
	hipDeviceSynchronize();

	stat = hipblasIsamax(handle, _wid*_hi, d_log_light, 1, &indexMax);
	stat = hipblasIsamin(handle, _wid*_hi, d_log_light, 1, &indexMin);

	lowpass <<< _bpg, _tpb >>> (d_log_light, d_log_light_lowpassed, run, _wid*_hi);
	hipDeviceSynchronize();

	if(run == 1)
		run = 0;

	float range = 8.0f;
	NormalizeRange2 <<< _bpg, _tpb >>> (d_log_light, d_log_light_lowpassed, range, d_log_normalized, indexMax-1, indexMin-1, _wid*_hi);
	hipDeviceSynchronize();
		    
	dim3 transpose_HtoV_grid3((_wid+BLOCK_DIM-1)/BLOCK_DIM, (_hi+BLOCK_DIM-1)/BLOCK_DIM), transpose_HtoV_threads3(BLOCK_DIM,BLOCK_DIM);
	dim3 transpose_VtoH_grid3((_hi+BLOCK_DIM-1)/BLOCK_DIM, (_wid+BLOCK_DIM-1)/BLOCK_DIM), transpose_VtoH_threads3(BLOCK_DIM,BLOCK_DIM);
	transpose <<<transpose_HtoV_grid3, transpose_HtoV_threads3 >>> (d_log_normalized_transposed, d_log_normalized, _wid, _hi, 1);
	gradient_vertical <<< _bpg, _tpb >>> (d_log_normalized_transposed, d_H, _hi, _wid, _wid*_hi);
	gradient_vertical <<< _bpg, _tpb >>> (d_log_normalized, d_V, _wid, _hi, _wid*_hi);
	hipDeviceSynchronize();   
    
	unsigned long detail_offset = 0;
	float s_scale = 0.25;
	float r_scale = 1.0;
	float local_sigS[NUM_DOMAINS] = {s_scale*20.0f, s_scale*50.0f, s_scale*100.0f};
	float local_sigR[NUM_DOMAINS] = {r_scale*0.33f, r_scale*0.67f, r_scale*1.34f};

	for(int j=0;j<NUM_DOMAINS;j++){
		cutilSafeCall(hipMemcpy(d_blur_transposed, d_log_normalized_transposed, _wid*_hi*sizeof(float), hipMemcpyDeviceToDevice));
		float _a0;
		unsigned int num_iterations = 2;
		for(int i = 0; i<num_iterations;i++){
			_a0 = exp( (-1 * sqrt(2)) / (local_sigS[j] * sqrt(3.0) * (powf(2, ( num_iterations-i-1 )) / sqrt( powf(4,num_iterations) - 1 ))) );
			domain_filter_vertical_RGB_noreg <<< (_hi+_tpb-1)/_tpb, _tpb >>> (d_blur_transposed, d_H, _a0, local_sigS[j]/local_sigR[j], _hi, _wid, 1);
			transpose <<<transpose_VtoH_grid3, transpose_VtoH_threads3>>> (d_blur, d_blur_transposed, _hi, _wid, 1);
			domain_filter_vertical_RGB_noreg <<< (_wid+_tpb-1)/_tpb, _tpb >>> (d_blur, d_V, _a0, local_sigS[j]/local_sigR[j], _wid, _hi, 1);
			hipDeviceSynchronize();
		}
		cutilSafeCall(hipMemcpy(d_detail+detail_offset, d_blur, _wid*_hi*sizeof(float), hipMemcpyDeviceToDevice));
		detail_offset += _wid*_hi;
	}

	extract_detail <<< _bpg, _tpb >>> (d_log_normalized, d_detail, d_detail_domains, NUM_DOMAINS, _wid*_hi);
	hipDeviceSynchronize();
		
	hipblasSasum(handle, _wid*_hi, d_blur, 1, &img_mean);

	CompressionAndSaturation <<< _bpg, _tpb >>> (d_light, d_detail_domains, d_blur, img_mean/(float)(_wid*_hi), d_output, _wid*_hi);

	cutStopTimer(&htimer);     
	avgHDR_HE += cutGetTimerValue(&htimer);
	cutResetTimer(&htimer);
	
	return 0;
}

void freeMem(){
#if BENCHMARK
	printf ("AVG ms time: %f ms\n", avgHDR_HE/cnt);
	printf ("AVG FPS: %f\n", 1000.0f/(avgHDR_HE/cnt));
#endif
	cutDeleteTimer(&htimer);
	
	hipFree(d_blur);
	hipFree(d_blur_transposed);
	
	hipFree(d_in);
	hipFree(d_light);
	hipFree(d_log_light);
	hipFree(d_log_light_lowpassed);
	hipFree(d_log_normalized);
	hipFree(d_log_normalized_transposed);
	hipFree(d_H);
	hipFree(d_V);

	hipFree(d_detail);
	hipFree(d_detail_domains);

	hipFree(d_output);
	hipFree(d_res);
	hipFree(d_weight);
	
	free(h_log_light);
	hipblasDestroy(handle);
	//hipDeviceReset();
}
